#include "hip/hip_runtime.h"
﻿#include "opencv2/imgproc/imgproc.hpp"
#include <opencv2/highgui.hpp>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE 16
#define KERNEL_SIZE 3             

using namespace std;

__global__ void laplacianFilter(unsigned char *srcImage, unsigned char *dstImage, unsigned int width, unsigned int height)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;

   //float kernel[3][3] = {0, -1, 0, -1, 4, -1, 0, -1, 0};
   //float kernel[3][3] = { 0, 1, 0, 1, -4, 1, 0, 1, 0 };
   float kernel[3][3] = {1, 4, 1, 4, -20, 4, 1, 4, 1};
   //float kernel[3][3] = {-1, -1, -1, -1, 8, -1, -1, -1, -1};   
   if((x >= KERNEL_SIZE /2) && (x < (width - KERNEL_SIZE /2)) && (y >= KERNEL_SIZE /2) && (y < (height - KERNEL_SIZE /2)))
   {
         float sum = 0;
         for(int ky = -KERNEL_SIZE / 2; ky <= KERNEL_SIZE / 2; ky++) {
            for(int kx = -KERNEL_SIZE / 2; kx <= KERNEL_SIZE / 2; kx++) {
               float src = srcImage[((y + ky) * width + (x + kx))]; 
               sum += src * kernel[ky + KERNEL_SIZE / 2][kx + KERNEL_SIZE / 2];
            }
         }
         dstImage[(y * width + x)] =  sum;
   }
}


void laplacianFilter_GPU_wrapper(const cv::Mat& input, cv::Mat& output)
{
        const int inputSize = input.cols * input.rows;
        const int outputSize = output.cols * output.rows;
        unsigned char *d_input, *d_output;
        
        hipMalloc<unsigned char>(&d_input, inputSize);
        hipMalloc<unsigned char>(&d_output, outputSize);

        hipMemcpy(d_input, input.ptr(), inputSize, hipMemcpyHostToDevice);
        const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        laplacianFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows);
        hipEventRecord(stop);

        hipMemcpy(output.ptr(), d_output, outputSize, hipMemcpyDeviceToHost);
        hipFree(d_input);
        hipFree(d_output);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        cout<< "\nTime in miliseconds: " << milliseconds << "\n";
}

int main(int argc, char** argv) {

    string input_file = "test.jpg";
    cv::Mat srcImage = cv::imread(input_file, cv::ImreadModes::IMREAD_UNCHANGED);
    if (srcImage.empty())
    {
        std::cout << "no image";
        return -1;
    }

    cv::cvtColor(srcImage, srcImage, cv::COLOR_BGR2GRAY);
    cv::Mat dstImage(srcImage.size(), srcImage.type());
    laplacianFilter_GPU_wrapper(srcImage, dstImage);
    imwrite("output3.jpg", dstImage);

    return 0;
}